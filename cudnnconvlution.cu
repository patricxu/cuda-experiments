#include <string>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hipDNN.h>

using namespace cv;
int main(int argc, char** argv)
{
	Mat img = imread("D:\\selfworkspace\\cudnndemo\\test1.jpg");
    Mat img_float;
    img.convertTo(img_float, CV_32F);

    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    hipdnnTensorDescriptor_t input_desc;
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT,
        1, img_float.channels(), img_float.rows, img_float.cols);

    hipdnnTensorDescriptor_t output_desc;
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT,
        1, img_float.channels(), img_float.rows, img_float.cols);

    int kernelShape[] = {3,3,3,3};
    Mat kernel = Mat(4, kernelShape, CV_32F, Scalar::all(0));
    float kernelMap[9] = {2,1,0,1,0,-1,0,-1,-2};
    for(int i = 0; i < 9; i++) {
        memcpy(kernel.data + i *9*sizeof(float), kernelMap, 9 * sizeof(float));
    }

    hipdnnFilterDescriptor_t kernel_desc;
    hipdnnCreateFilterDescriptor(&kernel_desc);
    hipdnnSetFilter4dDescriptor(kernel_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
            kernelShape[0], kernelShape[1], kernelShape[2], kernelShape[3]);

    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, 1,1,1,1,1,1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    size_t space_size = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, kernel_desc, conv_desc, output_desc, HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD, &space_size);
    void *workspace = nullptr;
    hipMalloc(&workspace, space_size);

    auto alpha = 1.0f;
    auto beta = 0.0f;
    size_t fm_size = img_float.channels() * img_float.rows * img_float.cols*sizeof(float);
    size_t wt_size = kernelShape[0] * kernelShape[1] * kernelShape[2] * kernelShape[3] * sizeof(float);
    void *dev_input = nullptr;
    hipMalloc(&dev_input, fm_size);
    hipMemcpy(dev_input, img_float.data, fm_size, hipMemcpyHostToDevice);
    void *dev_kernel = nullptr;
    hipMalloc(&dev_kernel, wt_size);
    hipMemcpy(dev_kernel, kernel.data, wt_size, hipMemcpyHostToDevice);

    void *dev_output = nullptr;
    hipMalloc(&dev_output, fm_size);
    hipdnnConvolutionForward(handle, &alpha, input_desc, dev_input, kernel_desc, dev_kernel, 
        conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD, workspace,space_size,&beta, output_desc, dev_output);

    Mat output(img_float);
    hipMemcpy(output.data, dev_output, fm_size, hipMemcpyDeviceToHost);
    Mat img_output;
    output.convertTo(img_output, CV_8UC3);
    hipFree(workspace);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(kernel_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(handle);

    imshow("output", img_output);
    waitKey(0);
    destroyWindow("output");
    return 0;
}