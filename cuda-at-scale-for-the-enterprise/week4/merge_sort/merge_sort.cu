#include "hip/hip_runtime.h"
#include "merge_sort.h"
#include <iostream>
// #include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>


#define min(a, b) (a < b ? a : b)
// Based on https://github.com/kevin-albert/cuda-mergesort/blob/master/mergesort.cu

__host__ std::tuple<dim3, dim3, int> parseCommandLineArguments(int argc, char** argv) 
{
    int numElements = 32;
    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    for (int i = 1; i < argc; i++) {
        if (argv[i][0] == '-' && argv[i][1] && !argv[i][2]) {
            char arg = argv[i][1];
            unsigned int* toSet = 0;
            switch(arg) {
                case 'x':
                    toSet = &threadsPerBlock.x;
                    break;
                case 'y':
                    toSet = &threadsPerBlock.y;
                    break;
                case 'z':
                    toSet = &threadsPerBlock.z;
                    break;
                case 'X':
                    toSet = &blocksPerGrid.x;
                    break;
                case 'Y':
                    toSet = &blocksPerGrid.y;
                    break;
                case 'Z':
                    toSet = &blocksPerGrid.z;
                    break;
                case 'n':
                    i++;
                    numElements = std::stoi(argv[i]);
                    break;
            }
            if (toSet) {
                i++;
                *toSet = (unsigned int) strtol(argv[i], 0, 10);
            }
        }
    }
    return {threadsPerBlock, blocksPerGrid, numElements};
}

__host__ long *generateRandomLongArray(int numElements)
{
    //TODO generate random array of long integers of size numElements
    long *randomLongs;
    randomLongs = (long*)malloc(numElements * sizeof(long));

    for(int i = 0; i < numElements; i++)
    {
        randomLongs[i] = rand() % 1000;
    }

    return randomLongs;
}

__host__ void printHostMemory(long *host_mem, int num_elments)
{
    // Output results
    for(int i = 0; i < num_elments; i++)
    {
        printf("%ld ",host_mem[i]);
    }
    printf("\n");
}

__host__ int main(int argc, char** argv) 
{

    auto[threadsPerBlock, blocksPerGrid, numElements] = parseCommandLineArguments(argc, argv);

    long *data = generateRandomLongArray(numElements);

    printf("Unsorted data: ");
    printHostMemory(data, numElements);

    data = mergesort(data, numElements, threadsPerBlock, blocksPerGrid);

    printf("Sorted data: ");
    printHostMemory(data, numElements);
}

__host__ std::tuple <long* ,long* ,dim3* ,dim3*> allocateMemory(long* data, int numElements, dim3* threadsDim, dim3* blocksDim)
{
    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long *D_data, *D_swp;
    dim3 *D_threads, *D_blocks;

    // Actually allocate the two arrays
    hipMalloc(&D_data, numElements * sizeof(long));
    hipMalloc(&D_swp, numElements * sizeof(long));
    hipMalloc(&D_threads, sizeof(dim3));
    hipMalloc(&D_blocks, sizeof(dim3));

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, numElements * sizeof(long), hipMemcpyHostToDevice);

    // Copy the thread / block info to the GPU as well
    hipMemcpy(D_threads, threadsDim, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, blocksDim, sizeof(dim3), hipMemcpyHostToDevice);

    return {D_data, D_swp, D_threads, D_blocks};
}

__host__ long* mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid)
 {
    auto[D_data, D_swp, D_threads, D_blocks] = allocateMemory(data, size, &threadsPerBlock, &blocksPerGrid);
    long* tmp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    // TODO Initialize timing metrics variable(s). The implementation of this is up to you
    

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //

    // printf("size %ld nThreads %ld\n", size, nThreads);
    int i = 0;
    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(D_data, D_swp, size, width, slices, D_threads, D_blocks); //TODO You will need to populate arguments for the kernel
        hipDeviceSynchronize();

        // Switch the input / output arrays instead of copying them around
        tmp = D_data;
        D_data = D_swp;
        D_swp = tmp;

        i++;
        // if(width==2)
        // break;
    }

    hipDeviceSynchronize();
    if(i%2 == 0)
        hipMemcpy(data, D_swp, size * sizeof(long), hipMemcpyDeviceToHost);
    else
        hipMemcpy(data, D_data, size * sizeof(long), hipMemcpyDeviceToHost);
    // TODO calculate and print to stdout kernel execution time

    // Free the GPU memory
    hipFree(D_data);
    hipFree(D_swp);
    hipFree(D_blocks);
    hipFree(D_threads);
    return data;
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);

    // TODO initialize 3 long variables start, middle, and end
    // middle and end do not have values set,
    // while start is set to the width of the merge sort data span * the thread index * number of slices that this kernel will sort
    long start, middle, end;
    start = width * idx * slices;
    for (long slice = 0; slice < slices; slice++) {
        // Break from loop when the start variable is >= size of the input array
        if(start >= size)
            break;

        // Set middle to be minimum middle index (start index plus 1/2 width) and the size of the input array
        middle = min((start + (width >> 1)), size);
        // Set end to the minimum of the end index (start index plus the width of the current data window) and the size of the input array
        end = min((start + width), size);

        // printf("start %ld, middle %ld, end %ld, size %ld\n", start, middle, end, size);
        // Perform bottom up merege given the two available arrays and the start, middle, and end variables
        gpu_bottomUpMerge(source, dest, start, middle, end);
        // Increase the start index by the width of the current data window
        start += width;
    }
}

//
// Finally, sort something gets called by gpu_mergesort() for each slice
// Note that the pseudocode below is not necessarily 100% complete you may want to review the merge sort algorithm.
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;

    // Create a for look that iterates between the start and end indexes
    for (int k = start; k < end; k++) {
        // if i is before the middle index and (j is the final index or the value at i <  the value at j)
        if (i < middle && (j >= end || source[i] < source[j]) ) {
            // set the value in the destination array at index k to the value at index i in the source array
            dest[k] = source[i];
            // increment i
            i++;
        } else {
            // set the value in the destination array at index k to the value at index j in the source array
            dest[k] = source[j];
            // increment j
            j++;
        }
    }
}