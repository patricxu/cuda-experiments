#include "hip/hip_runtime.h"
#include "memory_analysis.h"

__global__ void globalMemorySearch(int *input, int *indicator, int *totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = numElements / numThreads;

    if (threadId < numThreads)
    {
        // Create a for loop that handles the fact that each thread needs to search for n values in input
        // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
        for(int i = 0; i < elementsPerThread; i++)
        {
            if(input[threadId + i] == constant_search_value)
            {
                indicator[threadId + i] = 1;
                atomicAdd(totalFound, 1);
                // printf("found by thread %d at %d\n", threadId, i);
            }
            else
                indicator[threadId + i] = 0;
        }
    }
}

__global__ void sharedMemorySearch(int *input, int *indicator, int *totalFound, int numElements, int numThreads)
{
    extern __shared__ int sharedInput[];
    extern __shared__ int sharedOutput[];
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = numElements / numThreads;

    if (threadId < numThreads)
    {
        for(int i = 0; i < elementsPerThread; i++){
            sharedInput[threadId + i] = input[threadId + i];
        }

        __syncthreads();

        for(int i = 0; i < elementsPerThread; i++){
            if(sharedInput[threadId + i] == constant_search_value)
            {
                sharedOutput[threadId + i] = 1;
                atomicAdd(totalFound, 1);
            }
            else
                sharedOutput[threadId + i] = 0;
        }

        __syncthreads();

        for(int i = 0; i < elementsPerThread; i++){
            indicator[threadId + i] = sharedOutput[threadId + i];
        }        
    }
}

__global__ void constantMemorySearch(int *indicator, int *totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = numElements / numThreads;

    if (threadId < numThreads)
    {
        for(int i = 0; i < elementsPerThread; i++)
        {
            if(constant_input[threadId + i] == constant_search_value)
            {
                indicator[threadId + i] = 1;
                atomicAdd(totalFound, 1);
            }
            else
                indicator[threadId + i] = 0;
        }
    }
}

__global__ void registerMemorySearch(int *input, int *indicator, int *totalFound, int numElements, int numThreads)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = numElements / numThreads;

    if (threadId < numThreads)
    {
        // Create a for loop that handles the fact that each thread needs to search for n values in input
        // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
        for(int i = 0; i < elementsPerThread; i++)
        {
            if(input[threadId + i] == constant_search_value)
            {
                indicator[threadId + i] = 1;
                atomicAdd(totalFound, 1);
                // printf("found by thread %d at %d\n", threadId, i);
            }
            else
                indicator[threadId + i] = 0;
        }
    }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
__host__ int * allocatePageableRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);

    // Allocate the host pinned memory input pointer B
    int *data;
    hipHostAlloc((void**)&data, size, hipHostMallocDefault);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        data[i] = rand() % 255;
    }

    return data;
}

__host__ int * allocateDeviceMemory(int numElements)
{
    size_t size = numElements * sizeof(int);

    int *d_input = NULL;
    hipError_t err = hipMalloc(&d_input, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return d_input;
}

__host__ void copyFromHostToDevice(std::string kernelType, int *input, int numElements, int numThreads, int *d_input)
{
    size_t size = numElements * sizeof(int);

    if(!strcmp(kernelType.c_str(), "constant"))
    {
        // Copy input, numElements, threadSpan, and numThreads to constant memory
        int threadSpan = numElements / numThreads;
    } 
    else 
    {
        hipError_t err = hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
        // Copy input to d_input
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy array input from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
}

__host__ void executeKernel(int *d_input, int *d_indicator, int * d_totalFound, int numElements, int threadsPerBlock, std::string kernelType, int numThreads)
{
    // Launch the search CUDA Kernel
    if (!strcmp(kernelType.c_str(), "global"))
    {
        globalMemorySearch<<<1,threadsPerBlock>>>(d_input, d_indicator, d_totalFound, numElements, numThreads); // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "constant"))
    {
        constantMemorySearch<<<1,threadsPerBlock>>>(d_indicator, d_totalFound, numElements, numThreads);  // you will need to fill in function arguments appropriately
    } else if (!strcmp(kernelType.c_str(), "shared"))
    {
        unsigned int_array_size = numElements * sizeof(int);
        sharedMemorySearch<<<1,threadsPerBlock, int_array_size>>>(d_input, d_indicator, d_totalFound, numElements, numThreads); // you will need to fill in function arguments appropriately
    } else {
        globalMemorySearch<<<1,threadsPerBlock>>>(d_input, d_indicator, d_totalFound, numElements, numThreads); // you will need to fill in function arguments appropriately
        // registerMemorySearch<<<1,threadsPerBlock>>>();  // you will need to fill in function arguments appropriately
    }
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch %s kernel (error code %s)!\n", kernelType.c_str(), hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();
}

// Free device global memory
__host__ void deallocateMemory(int *d_input)
{

    hipError_t err = hipFree(d_input);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__host__ std::tuple<int, std::string, int, std::string> parseCommandLineArguments(int argc, char *argv[])
{
    int elementsPerThread = 2;
    int threadsPerBlock = 256;
    std::string currentPartId = "test";
    std::string kernelType = "global";

    for(int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if(option.compare("-t") == 0) 
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if(option.compare("-m") == 0) 
        {
            elementsPerThread = atoi(value.c_str());
        }
        else if(option.compare("-p") == 0) 
        {
            currentPartId = value;
        }
        else if(option.compare("-k") == 0) 
        {
            kernelType = value;
        }
    }

    return {elementsPerThread, currentPartId, threadsPerBlock, kernelType};
}

__host__ int * setUpInput(int numElements)
{
    srand(time(0));
    int *input;

    int searchValue = rand() % 255;
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constant_search_value), &searchValue, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    input = allocatePageableRandomHostMemory(numElements);

    err = hipMemcpyToSymbol(HIP_SYMBOL(constant_input), input, sizeof(int) * numElements, 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return input;
}

/*
 * Host main routine
 * -m elementsPerThread - the number of elements that a thread will search for a random value in
 * -p currentPartId - the Coursera Part ID
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 * -k the kernel type - global, constant, shared, register
 */
int main(int argc, char *argv[])
{
    auto[elementsPerThread, currentPartId, threadsPerBlock, kernelType] = parseCommandLineArguments(argc, argv);

    int numElements = elementsPerThread * threadsPerBlock;

    int *input = setUpInput(numElements);
    int *d_input = allocateDeviceMemory(numElements);
    int *d_indcator = allocateDeviceMemory(numElements);
    int *d_totalFound = allocateDeviceMemory(sizeof(int));
    int h_totalFound = 0;

    copyFromHostToDevice(kernelType, input, numElements, threadsPerBlock, d_input);

    // Start time including kernel processing time
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start,0);

    executeKernel(d_input, d_indcator, d_totalFound, numElements, threadsPerBlock, kernelType, threadsPerBlock);
    hipMemcpy(&h_totalFound, d_totalFound, sizeof(int), hipMemcpyDeviceToHost);
    // printf("total found %d\n", h_totalFound);
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start,stop);

    deallocateMemory(d_input);
    deallocateMemory(d_indcator);
    deallocateMemory(d_totalFound);
    cleanUpDevice();
    
    ofstream outputfile;
    outputfile.open ("output.csv", std::ios_base::app);
    outputfile << currentPartId.c_str() << "," << kernelType.c_str() << "," << threadsPerBlock << "," << elementsPerThread << "," << elapsedTime << "\n";
    outputfile.close();

    return 0;
}