#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Dimensions for matrix A (2x3), B (3x2), and hence C will be (2x2)
    int m = 2, n = 2, k = 3;
    float alpha = 1.0f, beta = 0.0f;

    // Define matrices A and B in row-major order
    float h_A[] = {1.0, 2.0, 3.0, // Matrix A
                   4.0, 5.0, 6.0};
    float h_B[] = {9.0, 8.0,       // Matrix B
                   7.0, 6.0,
                   5.0, 4.0};
    float h_C[4]; // Result matrix C will be stored here

    // Allocate device memory for matrices A, B, and C
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m*k*sizeof(float));
    hipMalloc(&d_B, k*n*sizeof(float));
    hipMalloc(&d_C, m*n*sizeof(float));

    // Copy matrices from the host to the device
    hipblasSetMatrix(m, k, sizeof(float), h_A, m, d_A, m);
    hipblasSetMatrix(k, n, sizeof(float), h_B, k, d_B, k);

    // Perform matrix multiplication with B transposed
    // Note that since B is being transposed, its dimensions are effectively swapped for the operation,
    // so we pass n as the leading dimension of B in cublasSgemm
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, d_A, k, d_B, n, &beta, d_C, m);

    // Copy the result back to the host memory
    hipblasGetMatrix(m, n, sizeof(float), d_C, m, h_C, m);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    // Print the result matrix C
    std::cout << "Result matrix C:" << std::endl;
    for(int i = 0; i < m*n; ++i) {
        std::cout << h_C[i] << " ";
        if((i + 1) % n == 0) std::cout << std::endl;
    }

    return 0;
}
