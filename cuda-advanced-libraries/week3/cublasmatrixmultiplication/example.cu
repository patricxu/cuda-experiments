
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cmath>
#include <sys/stat.h>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime_api.h>
// #include <cudnn.h>
#include <hipblas.h>
#include <memory>
#include <string.h>
#include <cstdint>
 
#define M 2
#define N 4
#define K 3
 
void printMatrix2(float** matrix, int row, int col) {
    for(int i=0;i<row;i++)
    {
        std::cout << std::endl;
        std::cout << " [ ";
        for (int j=0; j<col; j++) {
         std::cout << matrix[i][j] << " ";
        }
        std::cout << " ] ";
    }
    std::cout << std::endl;
}
 
int main(void)
{
        float alpha=1.0;
        float beta=0.0;
        float h_A[M][K]={ {1,2,3}, {4,5,6} };
        float h_B[K][N]={ {1,2,3,4}, {5,6,7,8}, {9,10,11,12} };
        float h_C[M][N] = {0};
        float *d_a,*d_b,*d_c;
        hipMalloc((void**)&d_a,M*K*sizeof(float));
        hipMalloc((void**)&d_b,K*N*sizeof(float));
        hipMalloc((void**)&d_c,M*N*sizeof(float));
        hipMemcpy(d_a,&h_A,M*K*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_b,&h_B,K*N*sizeof(float),hipMemcpyHostToDevice);
        hipMemset(d_c,0,M*N*sizeof(float));
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipblasSgemm(
            handle,
            HIPBLAS_OP_T,HIPBLAS_OP_T, 
            M, N, K,
            &alpha, 
            d_a, K, 
            d_b, N,
            &beta, 
            d_c, M);
        hipMemcpy(h_C,d_c,M*N*sizeof(float),hipMemcpyDeviceToHost);//此处的h_C是按列存储的C
        // printMatrix2((float**)h_C, N, M);//按行优先N行M列的顺序读取h_C相当于做了CT的结果
        return 0;
}