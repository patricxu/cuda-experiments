#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//http://www.techdarting.com/2014/03/matrix-multiplication-in-cuda-using.html


// This code assumes that your device support block size of 1024
#define MAX_RANGE 9999

const unsigned int TILE_WIDTH = 32;


#define gpu_errchk(ans) { gpu_assert((ans), __FILE__, __LINE__); }

inline void gpu_assert(hipError_t code, const char *file, int line,
                       bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "gpu_assert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        exit(code);
    }
}

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
    __shared__ float sA[TILE_WIDTH][TILE_WIDTH];   // Tile size of 32x32
    __shared__ float sB[TILE_WIDTH][TILE_WIDTH];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int ph = 0; ph < (((numAColumns - 1) / TILE_WIDTH) + 1); ph++) {
        if ((Row < numARows) && (threadIdx.x + (ph * TILE_WIDTH)) < numAColumns) {
            sA[threadIdx.y][threadIdx.x] = A[(Row * numAColumns) + threadIdx.x + (ph * TILE_WIDTH)];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (Col < numBColumns && (threadIdx.y + ph * TILE_WIDTH) < numBRows) {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + ph * TILE_WIDTH) * numBColumns + Col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; ++j) {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns) {
        C[Row * numCColumns + Col] = Cvalue;
    }
}

void matMultiplyOnHost(float *A, float *B, float *C, int numARows,
                       int numAColumns, int numBRows, int numBColumns,
                       int numCRows, int numCColumns) {
    for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++) {
            C[i * numCColumns + j] = 0.0;
            for (int k = 0; k < numCColumns; k++) {
                C[i * numCColumns + j] += A[i * numAColumns + k] * B[k * numBColumns + j];
            }
        }
    }
    return;
}

int main(int argc, char **argv) {
    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *hostComputedC;
    float *deviceA;
    float *deviceB;
    float *deviceC;

    // Please adjust rows and columns according to you need.
    int numARows = 512; // number of rows in the matrix A
    int numAColumns = 512; // number of columns in the matrix A
    int numBRows = 512; // number of rows in the matrix B
    int numBColumns = 512; // number of columns in the matrix B

    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    hostA = (float *) malloc(sizeof(float) * numARows * numAColumns);
    hostB = (float *) malloc(sizeof(float) * numBRows * numBColumns);

    for (int i = 0; i < numARows * numAColumns; i++) {
        hostA[i] = (rand() % MAX_RANGE) / 2.0;
    }
    for (int i = 0; i < numBRows * numBColumns; i++) {
        hostB[i] = (rand() % MAX_RANGE) / 2.0;
    }

    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float) * numCRows * numCColumns);
    hostComputedC = (float *) malloc(sizeof(float) * numCRows * numCColumns);

    // Allocating GPU memory
    gpu_errchk(hipMalloc((void **) &deviceA, sizeof(float) * numARows * numAColumns));
    gpu_errchk(hipMalloc((void **) &deviceB, sizeof(float) * numBRows * numBColumns));
    gpu_errchk(hipMalloc((void **) &deviceC, sizeof(float) * numCRows * numCColumns));

    // Copy memory to the GPU 
    gpu_errchk(hipMemcpy(deviceA, hostA, sizeof(float) * numARows * numAColumns, hipMemcpyHostToDevice));
    gpu_errchk(hipMemcpy(deviceB, hostB, sizeof(float) * numBRows * numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions 
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((numCColumns / TILE_WIDTH) + 1, (numCRows / TILE_WIDTH) + 1, 1);

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared <<<dimGrid, dimBlock>>>
                                       (deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipError_t err1 = hipPeekAtLastError();
    hipDeviceSynchronize();
    printf("Got CUDA error ... %s \n", hipGetErrorString(err1));

    // Copy the results in GPU memory back to the CPU    
    gpu_errchk(hipMemcpy(hostC, deviceC, sizeof(float) * numCRows * numCColumns, hipMemcpyDeviceToHost));

    matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    for (int i = 0; i < numCColumns * numCRows; i++) {
        if (hostComputedC[i] != hostC[i]) {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns,
                   i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }
    // Free the GPU memory
    gpu_errchk(hipFree(deviceA));
    gpu_errchk(hipFree(deviceB));
    gpu_errchk(hipFree(deviceC));

    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}